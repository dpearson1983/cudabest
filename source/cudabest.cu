#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <cmath>
#include <cstddef>
#include "../include/cudabest.hpp"

__constant__ double2 d_klim;
__constant__ double d_Deltak;
__constant__ int4 d_N;
__constant__ double3 d_kf

__device__ void swapIfGreater(double &a, double &b) {
    if (a > b) {
        double temp = a;
        a = b;
        b = temp;
    }
}

__device__ int getBispecBin(double k1, double k2, double k3, int numBins) {
    swapIfGreater(k1, k2);
    swapIfGreater(k1, k3);
    swapIfGreater(k2, k3);
    int i = (k1 - d_klim.x)/d_Deltak;
    int j = (k2 - d_klim.x)/d_Deltak;
    int k = (k3 - d_klim.x)/d_Deltak;
    int bin = k + numBins*(j + numBins*i);
    return bin;
}

__global__ void zeroArrays(hipfftDoubleComplex *dF0, hipfftDoubleComplex *dF2, hipfftDoubleComplex *dBij, 
                                     int4 N) {
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    
    if (tid < N.w) {
        this->d_F0[tid] = 0.0;
        this->d_F2[tid] = 0.0;
        this->d_Bij[tid] = 0.0;
    }
}

__global__ void calculateNumTriangles(int4 *d_kvecs, double *k_mags, unsigned long long int *dNtri, 
                                                int N_kvecs, int N_bins) {
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    int N_init = N_bins.w/blockDim.x + 1;
    int startInit = threadIdx.x*N_init;
    
    extern __shared__ unsigned long long int Ntri_local[];
    for (int i = startInit; i < startInit + N_init; ++i) {
        if (i < N_bins) {
            Ntri_local[i] = 0;
        }
    }
    __syncthreads();
    
    if (tid < N_kvecs) {
        int4 k_1 = d_kvecs[tid];
        double k_1mag = k_mags[tid];
        for (int i = tid; i < N_kvecs; ++i) {
            int4 k_2 = d_kvecs[i];
            double k_2mag = k_mags[i];
            int4 k_3 = {-k_1.x - k_2.x, -k_1.y - k_2.y, -k_1.z - k_2.z, 0};
            double3 k3 = {k_3.x*d_kf.x, k_3.y*d_kf.y, k_3.z*d_kf.z};
            double k_3mag = __dsqrt_rn(k3.x*k3.x + k3.y*k3.y + k3.z*k3.z);
            if (k_3mag >= d_klim.x && k_3mag < d_klim.y) {
                getBispecBin(k_1mag, k2_mag, k3_mag, N_bins);
                atomicAdd(&Ntri_local[bin], 1L);
            }
        }
    }
    __syncthreads();
    
    for (int i = startInit; i < startInit + N_init; ++i) {
        atomicAdd(dNtri[i], Ntri_local[i]);
    }
}

// __global__ void calculateBispectrum(int 4, double *B0, double *B2) {
//     int tid = threadIdx.x + blockDim.x*blockIdx.x;
// }
// 
// __global__ void bin(double3 *pos, double3 r_min, double3 Delta_r) {
//     int tid = threadIdx.x + blockDim.x*blockIdx.x;
// }

cudabest::cudabest(int Nx, int Ny, int Nz, double Lx, double Ly, double Lz, double x_min, double y_min, 
                   double z_min, double k_min, double k_max, int N_bins) {
    double2 k_lim = {k_min, k_max};
    this->k_f = {2.0*M_PI/Lx, 2.0*M_PI/Ly, 2.0*M_PI/L.z};
    this->N = {Nx, Ny, Nz, 0};
    this->N.w = Nx*Ny*Nz;
    this->L = {Lx, Ly, Lz};
    this->Delta_r = {Lx/Nx, Ly/Ny, Lz/Nz};
    this->r_min = {x_min, y_min, z_min};
    this->Delta_k = (k_max - k_min)/N_bins;
    
    hipMemcpyToSymbol(HIP_SYMBOL(d_klim), &k_lim, sizeof(double2));
    hipMemcpyToSymbol(HIP_SYMBOL(d_kf), this->k_f, sizeof(double3));
    hipMemcpyToSymbol(HIP_SYMBOL(d_N), this->N, sizeof(int4));
    hipMemcpyToSymbol(HIP_SYMBOL(d_Deltak), this->Delta_k, sizeof(double));
    
    hipMalloc((void **)&this->d_F0, this->N.w*sizeof(hipfftDoubleComplex));
    hipMalloc((void **)&this->d_F2, this->N.w*sizeof(hipfftDoubleComplex));
    hipMalloc((void **)&this->d_Bij, this->N.w*sizeof(hipfftDoubleComplex));
    
    cudabest::zeroArrays(this->d_F0, this->d_F2, this->d_Bij, this->N);
}

void cudabest::getBispectrum(std::vector<double3> &gals, std::vector<double3> &rans, std::vector<double> &B_0,
                             std::vector<double> &B_2) {
    std::cout << "Not implemented." << std::endl;
}
