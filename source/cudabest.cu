#include "hip/hip_runtime.h"
#include <vector>
#include <hip/hip_vector_types.h>
#include <hipfft/hipfft.h>
#include <cmath>
#include "../include/cudabest.h"

__device__ void cudabest::swapIfGreater(double &a, double &b) {
    if (a > b) {
        double temp = a;
        a = b;
        b = temp;
    }
}

__device__ int cudabest::getBispecBin(double k1, double k2, double k3, double Delta_k, int numBins, 
                                      double k_min) {
    cudabest::swapIfGreater(k1, k2);
    cudabest::swapIfGreater(k1, k3);
    cudabest::swapIfGreater(k2, k3);
    int i = (k1 - k_min)/Delta_k;
    int j = (k2 - k_min)/Delta_k;
    int k = (k3 - k_min)/Delta_k;
    int bin = k + numBins*(j + numBins*i);
    return bin;
}

__global__ void cudabest::zeroArrays(double *d_F0, double *d_F2, double *d_Bij, int4 N) {
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    
    if (tid < N.w) {
        d_F0[tid] = 0.0;
        d_F2[tid] = 0.0;
        d_Bij[tid] = 0.0;
    }
}

__global__ void cudabest::calculateNumTriangles(int4 *d_kvecs,double *k_mags, unsigned long long int *d_Ntri, 
                                                int N_kvecs, int4 N, double3 k_f, int4 N_bins) {
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    int N_init = N_bins.w/blockDim.x + 1;
    int startInit = threadIdx.x*N_init;
    
    extern __shared__ unsigned long long int Ntri_local[];
    for (int i = startInit; i < startInit + N_init; ++i) {
        if (i < N_bins.w) {
            Ntri_local[i] = 0;
        }
    }
    __syncthreads();
    
    if (tid < N_kvecs) {
        int4 k_1 = d_kvecs[tid];
    }
}
